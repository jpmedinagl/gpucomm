#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fcntl.h>
#include <unistd.h>

int main() {
    hipSetDevice(1);

    const int n = 256;

    int * recBuf;
    hipMalloc(&recBuf, n * sizeof(int));

    hipIpcMemHandle_t handle;
    hipIpcGetMemHandle(&handle, recBuf);

    int fd = open("ipc_handle.bin", O_CREAT | O_WRONLY, 0666);
    write(fd, &handle, sizeof(handle));
    close(fd);

    printf("GPU 1 ready\n");

    int * hostBuf = (int *)malloc(n * sizeof(int));

    while (1) {
        sleep(3);

        hipMemcpy(hostBuf, recBuf, n * sizeof(int), hipMemcpyDeviceToHost);
        
        printf("Received\n");
        for (int i = 0; i < n; i++) {
            printf("%d ", hostBuf[i]);
        }
        printf("\n");
        
        free(hostBuf);
        hipFree(recBuf);
        exit(1);
    }

    return 0;
}