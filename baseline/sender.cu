#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fcntl.h>
#include <unistd.h>

__global__ void fill(int *buf, size_t n) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        buf[i] = i;
}

__global__ void send(int *sendBuf, int *recBuf, size_t n) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        recBuf[i] = sendBuf[i];
}

int main() {
    hipSetDevice(0);

    const int n = 256;

    int * sendBuf;
    hipMalloc(&sendBuf, n * sizeof(int));

    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;
    fill<<<blocks, threadsPerBlock>>>(sendBuf, n);
    hipDeviceSynchronize();

    hipIpcMemHandle_t handle;
    int fd = open("ipc_handle.bin", O_RDONLY);
    read(fd, &handle, sizeof(handle));
    close(fd);

    int *recBuf;
    hipIpcOpenMemHandle((void**)&recBuf, handle, hipIpcMemLazyEnablePeerAccess);

    float elapsedTime = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    send<<<blocks, threadsPerBlock>>>(sendBuf, recBuf, n);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    printf("GPU 0 sent to receiver\n");

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Kernel send time: %.3f ms\n", elapsedTime);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipFree(sendBuf);
    hipIpcCloseMemHandle(recBuf);
    return 0;
}