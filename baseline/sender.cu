#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <fcntl.h>
#include <unistd.h>

__global__ void fill(int *buf, size_t n) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        buf[i] = i;
}

__global__ void send(int *sendBuf, int *recBuf, size_t n) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        recBuf[i] = sendBuf[i];
}

int main() {
    hipSetDevice(0);

    const int n = 256;

    int * sendBuf;
    hipMalloc(&sendBuf, n * sizeof(int));
    fill<<256, 1>>(sendBuf, n);
    hipDeviceSynchronize();

    hipIpcMemHandle_t handle;
    int fd = open("ipc_handle.bin", O_RDONLY);
    read(fd, &handle, sizeof(handle));
    close(fd);

    int *recBuf;
    hipIpcOpenMemHandle((void**)&recBuf, handle, hipIpcMemLazyEnablePeerAccess);


    send<<256, 1>>(sendBuf, recBuf, n);
    hipDeviceSynchronize();

    printf("GPU 0 sent to receiver\n");

    hipFree(sendBuf);
    hipIpcCloseMemHandle(recBuf);
    return 0;
}