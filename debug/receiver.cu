#include "receiver.cuh"

void Receiver::send_addr(int sockfd)
{
    // 1. send key
    size_t rkey_size;
    void *rkey_buffer;
    UCS_CHECK(ucp_rkey_pack(context, memh, &rkey_buffer, &rkey_size));
    
    printf("rkey send: %p %zu\n\n", rkey_buffer, rkey_size);

    socket_send(sockfd, &rkey_size, sizeof(rkey_size));
    socket_send(sockfd, rkey_buffer, rkey_size);
    ucp_rkey_buffer_release(rkey_buffer);

    // 2. send ring buffer information
    socket_send(sockfd, &buffer, sizeof(buffer));

    printf("local info:\n");
    printf("    buf_ptr %p\n", (void*)&buffer);
    printf("    buf: %p\n", (void*)buffer);
}

Receiver::Receiver(ucp_context_h ctx, ucp_worker_h wrk, ucp_ep_h endpoint,
                   int sockfd)
    : context(ctx), worker(wrk), ep(endpoint)
{    
    uint64_t* remote_buffer;
    hipMalloc(&remote_buffer, sizeof(uint64_t));

    uint64_t init_value = 0xa;
    hipMemcpy(remote_buffer, &init_value, sizeof(uint64_t), hipMemcpyHostToDevice);

    ucp_mem_map_params_t mem_map_params = {
        .field_mask = UCP_MEM_MAP_PARAM_FIELD_ADDRESS |
                      UCP_MEM_MAP_PARAM_FIELD_LENGTH |
                      UCP_MEM_MAP_PARAM_FIELD_MEMORY_TYPE,
        .address = remote_buffer,
        .length = sizeof(uint64_t),
        .memory_type = UCS_MEMORY_TYPE_CUDA
    };
    
    ucp_mem_map(ucp_context, &mem_map_params, &memh);

    buffer = remote_buffer;

    send_addr(sockfd);
}

void Receiver::print_rb()
{
    // Print the values of rand_ptr and rand
    printf("rand: %p\n", rand);
}
