#include "utils.h"
#include "receiver.cuh"

int main() 
{
    ucp_context_h context;
    ucp_worker_h worker;
    ucp_ep_h ep;

    CUDA_CHECK(hipSetDevice(1));

    init(&context, &worker);
    
    // Socket setup
    int sockfd = socket(AF_INET, SOCK_STREAM, 0);

    struct sockaddr_in addr;
    memset(&addr, 0, sizeof(addr));
    addr.sin_family = AF_INET;
    addr.sin_port = htons(PORT);
    addr.sin_addr.s_addr = INADDR_ANY;
    
    int optval = 1;
    setsockopt(sockfd, SOL_SOCKET, SO_REUSEADDR, &optval, sizeof(optval));

    bind(sockfd, (struct sockaddr*)&addr, sizeof(addr));
    listen(sockfd, 1);
    sockfd = accept(sockfd, NULL, NULL);

    // exchange addresses + keys!

    printf("socked connected\n");

    create_ep(sockfd, worker, &ep);

    Receiver receiver(context, worker, ep, sockfd);

    printf("Receiver connected\n\n");

    while (1) {
        sleep(1);
        receiver.print_rb();
        // sleep(1);
    }

    // sleep(3);

    // for (int i = 0; i < NUM_CHUNKS; i++) {
    //     void * out_chunk;
    //     hipMalloc(&out_chunk, CHUNK_SIZE);

    //     receiver.dequeue(out_chunk);
    //     // receiver.print_rb();
    //     // sleep(1);

    //     // char host_data[CHUNK_SIZE + 1] = {0};
    //     // hipMemcpy(host_data, out_chunk, CHUNK_SIZE, hipMemcpyDeviceToHost);

    //     // printf("Received chunk %d: %s\n", i, host_data);

    //     hipFree(out_chunk);
    // }

    return 0;
}