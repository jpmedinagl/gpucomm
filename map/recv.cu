#include "hip/hip_runtime.h"
#include <cuco/static_map.cuh>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <iostream>

template <typename Map>
__global__ void find_in_map_kernel(Map map, int* queries, int* results, int num_queries) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_queries) {
        auto found = map.find(queries[idx]);

         if (found != map.end()) {
            // If the key is found, store the value
            results[idx] = found->second; // Assuming the iterator points to a pair {key, value}
        } else {
            // Key not found, handle as needed
            results[idx] = -1;
        }
    }
}

int main() {
    hipIpcMemHandle_t handle;
    int fd = open("ipc_handle.bin", O_RDONLY);
    read(fd, &handle, sizeof(handle));
    close(fd);

    auto find_ref;
    hipIpcOpenMemHandle((void**)&find_ref, handle, hipIpcMemLazyEnablePeerAccess);

    find_in_map_kernel<<<(queries.size() + 255) / 256, 256>>>(find_ref, d_queries, d_results, queries.size());

    hipMemcpy(thrust::raw_pointer_cast(results.data()), d_results, queries.size() * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Query results:\n";
    for (int i = 0; i < queries.size(); ++i) {
        std::cout << "Key " << queries[i] << " -> Value ";
        if (results[i] == -1) {
            std::cout << "NOT FOUND";
        } else {
            std::cout << results[i];
        }
        std::cout << "\n";
    }

    hipFree(d_queries);
    hipFree(d_results);

    return 0;
}